#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "UnixStopwatch.h"

__device__ int getThreadId(){
  int blockSize = blockDim.x * blockDim.y * blockDim.z;
    
  int ret = (blockIdx.x * (int) gridDim.y * blockSize) +
            (blockIdx.y * blockSize) +
            (threadIdx.x * blockDim.y * blockDim.z) +
            (threadIdx.y * blockDim.z);
  return ret;
}

__global__ void kernel(float * a, float * b, float * c, int size){

  int threadId = getThreadId();
  int i = threadId / size;
  int j = threadId % size;

  float sum = 0;
  for(int k = 0; k < size; ++k){
    sum += a[i*size+k] * b[k*size+j];
  }
  c[i*size+j] = sum;  
} 

void initAB(float * array, int size){
  int index = 0;
  for(int i = 0; i < size; ++i){
    for(int j = 0; j < size; ++j){
      array[index] = index % 8;
      ++index;
    }
  }
}

float * computeGold(float * a, float * b, int size){
  float * gold_c = (float *) malloc(size*size*sizeof(float));
  
  for(int i = 0; i < size; ++i){
    for(int j = 0; j < size; ++j){
      float sum = 0;
      for(int k = 0; k < size; ++k){
        sum += a[i*size+k] * b[k*size+j];
      }
      gold_c[i*size+j] = sum;
    }
  }
  return gold_c;
}

void checkResults(float * a, float * b, float * c, int size){
  float * gold_c = computeGold(a, b, size);
  int match = 1;
  for(int i = 0; i < size; ++i){
    for(int j = 0; j < size; ++j){
      float lhs = c[i*size+j];
      float rhs = gold_c[i*size+j];
      if(lhs != rhs){
        match = 0;
      }
    }
  }
  if(match){
    printf("  results match\n");
  } else {
    printf("  results mismatch\n");
  }
  free(gold_c);
}

int main(int argc, char * argv[]){
  int size = 2048;
  float * host_a = (float *) malloc(size*size*sizeof(float));
  float * host_b = (float *) malloc(size*size*sizeof(float));
  float * host_c = (float *) malloc(size*size*sizeof(float));

  float * device_a;
  float * device_b;
  float * device_c;

  initAB(host_a, size);
  initAB(host_b, size);  

  hipError_t error;
  error = hipMalloc((void **) &device_a, size*size*sizeof(float));
  if(error != hipSuccess){
    printf("hipMalloc returned error code %d, line(%d)\n", error, __LINE__);
    exit(0);
  }

  error = hipMalloc((void **) &device_b, size*size*sizeof(float));
  if(error != hipSuccess){
    printf("hipMalloc returned error code %d, line(%d)\n", error, __LINE__);
    exit(0);
  }

  error = hipMalloc((void **) &device_c, size*size*sizeof(float));
  if(error != hipSuccess){
    printf("hipMalloc returned error code %d, line(%d)\n", error, __LINE__);
    exit(0);
  }

  for(int i = 0; i < 8; ++i){
    UnixStopwatch watch;
    watch.start();
  
    error = hipMemcpy(device_a, host_a, size*size*sizeof(float), hipMemcpyHostToDevice);
    if(error != hipSuccess){
      printf("hipMemcpy returned error code %d, line(%d)\n", error, __LINE__);
      exit(0);
    }

    error = hipMemcpy(device_b, host_b, size*size*sizeof(float), hipMemcpyHostToDevice);
    if(error != hipSuccess){
      printf("hipMemcpy returned error code %d, line(%d)\n", error, __LINE__);
      exit(0);
    }

    kernel<<<16384, 256>>>(device_a, device_b, device_c, size);
    hipDeviceSynchronize();

    error = hipMemcpy(host_c, device_c, size*size*sizeof(float), hipMemcpyDeviceToHost);
    if(error != hipSuccess){
      printf("hipMemcpy returned error code %d, line(%d)\n", error, __LINE__);
      exit(0);
    }
    watch.stop();
    printf("time: %d\n", watch.getTime());

    checkResults(host_a, host_b, host_c, size);
  }
}
